/*
 * SPDX-FileCopyrightText: Copyright (c) 2023 NVIDIA CORPORATION & AFFILIATES. All rights reserved.
 * SPDX-License-Identifier: MIT
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include <hip/hip_fp16.h>

#include <algorithm>
#include <numeric>
#include <iostream>
#include "common/check.hpp"
#include "common/launch.cuh"
#include "common/tensorrt.hpp"
#include "point_encoder.hpp"

namespace bevfusion {
namespace point_encoder {

class PointEncoderImplement : public PointEncoder {
 public:
  virtual ~PointEncoderImplement() {
    if (output_) checkRuntime(hipFree(output_));
  }

  virtual bool init(const std::string& model) {
    engine_ = TensorRT::load(model);
    if (engine_ == nullptr) return false;

    if (engine_->has_dynamic_dim()) {
      printf("Dynamic shapes are not supported.\n");
      return false;
    }

    int output_binding = 1;
    auto shape = engine_->static_dims(output_binding);
    output_shape_ = engine_->static_dims(output_binding);
    Asserts(engine_->dtype(output_binding) == TensorRT::DType::HALF, "Invalid binding data type.");

    size_t volumn = std::accumulate(shape.begin(), shape.end(), 1, std::multiplies<int>());
    checkRuntime(hipMalloc(&output_, volumn * sizeof(half)));
    return true;
  }

  virtual void print() override { engine_->print("PointEncoder"); }

  virtual nvtype::half* forward(const nvtype::half* voxels, void* stream) override {
    hipStream_t _stream = static_cast<hipStream_t>(stream);
    engine_->forward({/* input  */ voxels,
                      /* output */ output_},
                     _stream);
    return output_;
  }

  virtual std::vector<int> point_feature_shape() override { return output_shape_; }

 private:
  std::shared_ptr<TensorRT::Engine> engine_;
  nvtype::half* output_ = nullptr;
  nvtype::half* spatial_feature = nullptr;
  std::vector<std::vector<int>> bindshape_;
  std::vector<int> output_shape_;
};

std::shared_ptr<PointEncoder> create_point_encoder(const std::string& param) {
  std::shared_ptr<PointEncoderImplement> instance(new PointEncoderImplement());
  if (!instance->init(param)) {
    instance.reset();
  }
  return instance;
}

};  // namespace point_encoder
};  // namespace bevfusion
